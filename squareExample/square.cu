
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 *
 * See cuda.h for error code descriptions.
 */
#define CHECK_CUDA_RESULT(N) {											\
	hipError_t result = N;												\
	if (result != 0) {													\
		printf("CUDA call on line %d returned error %d\n", __LINE__,	\
			result);													\
		exit(1);														\
	} }

/*Thread - take in a number and square it*/
__global__ void square(float * d_in, float * d_out)
{
	/*threadIdx is actually a C struct with three members
	 * x, y, z - we only need x right now*/
	int threadId = threadIdx.x;
	float data = d_in[threadId];
	d_out[threadId] = data * data;
}

int main(int argc, char **argv)
{
	const int ARRAY_SIZE = 64;
	const int ARRAY_BYTES = 64 * sizeof(float);

	/*Allocate CPU memory*/
	float * h_in = (float *) malloc(ARRAY_BYTES);
	float * h_out = (float *) malloc(ARRAY_BYTES);

	/*Declare GPU pointers*/
	float * d_in;
	float * d_out;

	for (int index = 0; index < ARRAY_SIZE; index++)
	{
		/*Fill in host array*/
		h_in[index] = float(index);
	}

	/*Allocate memory on the GPU*/
	hipMalloc((void **) &d_in, ARRAY_BYTES);
	hipMalloc((void **) &d_out, ARRAY_BYTES);

	/*Now memory is allocated and filled on CPU side, and allocated on GPU side
	 * Next step is to copy the input array from the CPU to the GPU*/

	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);	//Host is CPU, device is GPU

	/*Launch the GPU kernel
	 * 1 block
	 * ARRAY_SIZE threads in the block*/
	square<<<1, ARRAY_SIZE>>>(d_in, d_out);

	/*the kernel call is blocking?
	 * Anyways copy from d_out to h_out*/
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	/*On CPU, print results to ensure correctness*/

	for (int index = 0; index < ARRAY_SIZE; index++)
	{
		printf("Num: %f \t Num Squared: %f\n", h_in[index], h_out[index]);

	}

	/*Never forget to free the memory when you are done*/
	free(h_out);
	free(h_in);
	hipFree(d_out);
	hipFree(d_in);
	return 0;
}
